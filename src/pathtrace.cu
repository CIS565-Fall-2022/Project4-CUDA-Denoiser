#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#include ""

#define ERRORCHECK 1

#define gauss 1
#define stride 25
#define show_pos 0

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__global__ void gbufferToPBO_Normal(uchar4* pbo, glm::ivec2 resolution, GBufferPixel* gBuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        float timeToIntersect = gBuffer[index].t * 256.0;

        glm::vec3 nor = gBuffer[index].normal;
        pbo[index].w = 0;
        pbo[index].x = glm::clamp(abs((int)(nor.x * 255.f)), 0, 255);
        pbo[index].y = glm::clamp(abs((int)(nor.y * 255.f)), 0, 255);
        pbo[index].z = glm::clamp(abs((int)(nor.z * 255.f)), 0, 255);
    }
}

__global__ void gbufferToPBO_Position(uchar4* pbo, glm::ivec2 resolution, GBufferPixel* gBuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);

        glm::vec3 position = gBuffer[index].position;
        pbo[index].w = 0;
        pbo[index].x = glm::clamp(abs(position.x * stride), 0.f, 255.f);
        pbo[index].y = glm::clamp(abs(position.y * stride), 0.f, 255.f);
        pbo[index].z = glm::clamp(abs(position.z * stride), 0.f, 255.f);

    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static GBufferPixel* dev_gBuffer = NULL;
// TODO: static variables for device memory, any extra info you need, etc

static glm::vec3* dev_denoised_image = NULL;
static glm::vec3* dev_denoised_buffer = NULL;
static float* dev_gauss_kernal = NULL;

//https://www.geeksforgeeks.org/gaussian-filter-generation-c/
static float gauss_kernel[25] = {
0.00296902, 0.0133062, 0.0219382, 0.0133062, 0.00296902,
0.0133062, 0.0596343, 0.0983203, 0.0596343, 0.0133062,
0.0219382, 0.0983203, 0.162103, 0.0983203, 0.0219382,
0.0133062, 0.0596343, 0.0983203, 0.0596343, 0.0133062,
0.00296902, 0.0133062, 0.0219382, 0.0133062, 0.00296902,
};

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

    // TODO: initialize any extra device memeory you need
    hipMalloc(&dev_denoised_image, pixelcount * sizeof(glm::vec3));
    hipMalloc(&dev_denoised_buffer, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_gauss_kernal, 25 * sizeof(float));
    hipMemcpy(dev_gauss_kernal, gauss_kernel, 25 * sizeof(float), hipMemcpyHostToDevice);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    hipFree(dev_gBuffer);
    // TODO: clean up any extra device memory you created
    hipFree(dev_denoised_image);
    hipFree(dev_denoised_buffer);
    hipFree(dev_gauss_kernal);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void shadeSimpleMaterials (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment segment = pathSegments[idx];
    if (segment.remainingBounces == 0) {
      return;
    }

    if (intersection.t > 0.0f) { // if the intersection exists...
      segment.remainingBounces--;
      // Set up the RNG
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, segment.remainingBounces);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        segment.color *= (materialColor * material.emittance);
        segment.remainingBounces = 0;
      }
      else {
        segment.color *= materialColor;
        glm::vec3 intersectPos = intersection.t * segment.ray.direction + segment.ray.origin;
        scatterRay(segment, intersectPos, intersection.surfaceNormal, material, rng);
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      segment.color = glm::vec3(0.0f);
      segment.remainingBounces = 0;
    }

    pathSegments[idx] = segment;
  }
}

__global__ void generateGBuffer (
  int num_paths,
  ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
  GBufferPixel* gBuffer) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    gBuffer[idx].t = shadeableIntersections[idx].t;
    gBuffer[idx].normal = shadeableIntersections[idx].surfaceNormal;
    gBuffer[idx].position = getPointOnRay(pathSegments[idx].ray, shadeableIntersections[idx].t);

  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Pathtracing Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * NEW: For the first depth, generate geometry buffers (gbuffers)
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally:
    //     * if not denoising, add this iteration's results to the image
    //     * TODO: if denoising, run kernels that take both the raw pathtraced result and the gbuffer, and put the result in the "pbo" from opengl

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

    // Empty gbuffer
    hipMemset(dev_gBuffer, 0, pixelcount * sizeof(GBufferPixel));

	// clean shading chunks
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    bool iterationComplete = false;
	while (!iterationComplete) {

	    // tracing
	    dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
	    computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>> (
		    depth
		    , num_paths
		    , dev_paths
		    , dev_geoms
		    , hst_scene->geoms.size()
		    , dev_intersections
		    );
	    checkCUDAError("trace one bounce");
	    hipDeviceSynchronize();

        if (depth == 0) {
            generateGBuffer<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_intersections, dev_paths, dev_gBuffer);
        }

	    depth++;

        shadeSimpleMaterials<<<numblocksPathSegmentTracing, blockSize1d>>> (
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials
        );
        iterationComplete = depth == traceDepth;
	}

  // Assemble this iteration and apply it to the image
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // CHECKITOUT: use dev_image as reference if you want to implement saving denoised images.
    // Otherwise, screenshots are also acceptable.
    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}

__global__ void ATrousDenoise(float c_phi, float n_phi, float p_phi, glm::ivec2 resolution, int stepWidth, GBufferPixel* gBuffer,
                              glm::vec3* pt_image, glm::vec3* denoised_image)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x < resolution.x && y < resolution.y) {
        float cum_w = 0.0f;
        glm::vec3 sum(0.f, 0.f, 0.f);

        int index = y * resolution.x + x;
        glm::vec3 nval = gBuffer[index].normal;
        glm::vec3 pval = gBuffer[index].position;
        glm::vec3 cval = pt_image[index];

        float kernal[5] = { 0.0625, 0.25, 0.375, 0.25, 0.0625 };
        for (int i = 0; i < 5; i++) {
            for (int j = 0; j < 5; j++) {
                glm::ivec2 offset;
                offset.x = x + (i - 2) * stepWidth;
                offset.y = y + (j - 2) * stepWidth;
                offset = glm::clamp(offset, glm::ivec2(0, 0), glm::ivec2(resolution.x - 1, resolution.y - 1));

                int tmp = offset.y * resolution.x + offset.x;

                glm::vec3 ctmp = pt_image[tmp];

                glm::vec3 t = cval - ctmp;

                float dist2 =  glm::dot(t, t);
                float c_w = min(exp(-dist2 / (c_phi* c_phi)), 1.0f);

                glm::vec3 ntmp = gBuffer[tmp].normal;
                t = nval - ntmp;
                dist2 = max(glm::dot(t, t) / (stepWidth * stepWidth), 0.0f);
                float n_w = min(exp(-dist2 / (n_phi* n_phi)), 1.f);

                glm::vec3 ptmp = gBuffer[tmp].position;
                t = pval - ptmp;
                dist2 = glm::dot(t, t);
                float p_w = min(exp(-dist2 / (p_phi * n_phi)), 1.f);

                float filter = kernal[i] * kernal[j];//NOT SURE WHETHER THIS KERNAL IS CORREST
                float weight = c_w * n_w * p_w;
                sum += ctmp * weight * filter; 
                cum_w += weight * filter;

            }
        }
        denoised_image[index] = sum / cum_w;    
    }
}

__global__ void GaussBlur(glm::ivec2 resolution, int stepWidth, float* gauss_kernal,
    glm::vec3* pt_image, glm::vec3* denoised_image)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x < resolution.x && y < resolution.y) {
        int index = y * resolution.x + x;
        glm::vec3 sum(0.f, 0.f, 0.f);
        for (int i = 0; i < 5; i++) {
            for (int j = 0; j < 5; j++) {
                glm::ivec2 offset;
                offset.x = x + (i - 2) * stepWidth;
                offset.y = y + (j - 2) * stepWidth;
                offset = glm::clamp(offset, glm::ivec2(0, 0), glm::ivec2(resolution.x - 1, resolution.y - 1));

                int tmp = offset.y * resolution.x + offset.x;

                glm::vec3 ctmp = pt_image[tmp];
                float weight = gauss_kernal[j * 5 + i];
                sum += weight * ctmp;
            }
        }
        denoised_image[index] = sum;
    }
}

void Denoise_Image(float c_phi, float n_phi, float p_phi, float stepWidth)
{
    Camera& cam = hst_scene->state.camera;
    glm::ivec2 resolution = cam.resolution;

    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    hipMemcpy(dev_denoised_image, dev_image, resolution.x * resolution.y * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

    int iteration = ceil(glm::log2(stepWidth));
    for (int i = 0; i < iteration; i++) {
#if gauss
        GaussBlur << <blocksPerGrid2d, blockSize2d >> > (resolution, 1 << i, dev_gauss_kernal, dev_denoised_image, dev_denoised_buffer);
#else 
        ATrousDenoise << <blocksPerGrid2d, blockSize2d >> > (c_phi, n_phi, p_phi, resolution, 1 << i, dev_gBuffer, dev_denoised_image, dev_denoised_buffer);
        
#endif
        std::swap(dev_denoised_buffer, dev_denoised_image);
    }
    hipMemcpy(hst_scene->state.image.data(), dev_denoised_image, resolution.x * resolution.y * sizeof(glm::vec3), hipMemcpyDeviceToHost);
}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something that you can visualize for debugging.
void showGBuffer(uchar4* pbo) {
    const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
#if show_pos
    gbufferToPBO_Position << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, dev_gBuffer);

#else
    gbufferToPBO_Normal <<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, dev_gBuffer);

#endif
}

void showImage(uchar4* pbo, int iter) {
const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
}

void showDenoisedImage(uchar4* pbo, int iter) {
    const Camera& cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_denoised_image);
}
