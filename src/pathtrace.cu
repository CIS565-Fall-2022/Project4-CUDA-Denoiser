#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "main.h"
#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1
#define DENOISE 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (hipSuccess == err) {
    return;
  }

  fprintf(stderr, "CUDA error");
  if (file) {
    fprintf(stderr, " (%s:%d)", file, line);
  }
  fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
  getchar();
#  endif
  exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
  int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
  return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
  int iter, glm::vec3* image) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < resolution.x && y < resolution.y) {
    int index = x + (y * resolution.x);
    glm::vec3 pix = image[index];

    glm::ivec3 color;
    color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
    color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
    color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

    // Each thread writes one pixel location in the texture (textel)
    pbo[index].w = 0;
    pbo[index].x = color.x;
    pbo[index].y = color.y;
    pbo[index].z = color.z;
  }
}

__global__ void gbufferToPBO(uchar4* pbo, glm::vec3* image, glm::ivec2 resolution, GBufferPixel* gBuffer, int type) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < resolution.x && y < resolution.y) {
    int index = x + (y * resolution.x);
    float timeToIntersect = gBuffer[index].t * 256.0;

    glm::vec3 output;
    if (type == 0) {
      output.x = glm::abs(gBuffer[index].normal.x * 255.f);
      output.y = glm::abs(gBuffer[index].normal.y * 255.f);
      output.z = glm::abs(gBuffer[index].normal.z * 255.f);
    }
    else if (type == 1) {
      output = gBuffer[index].pos * 256.f * 0.1f;
    }
    else {
      output = glm::vec3(gBuffer[index].t) * 256.f;
    }

    pbo[index].w = 0;
    pbo[index].x = output.x;
    pbo[index].y = output.y;
    pbo[index].z = output.z;

    image[index].x = output.x;
    image[index].y = output.y;
    image[index].z = output.z;
  }
}

static Scene* hst_scene = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static GBufferPixel* dev_gBuffer = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

glm::vec3* dev_image_denoise = NULL;
glm::vec3* dev_image_denoise_tmp = NULL;
glm::vec3* dev_image_gBuffer = NULL;

void pathtraceInit(Scene* scene) {
  hst_scene = scene;
  const Camera& cam = hst_scene->state.camera;
  const int pixelcount = cam.resolution.x * cam.resolution.y;

  hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
  hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

  hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

  // TODO: initialize any extra device memeory you need

  hipMalloc(&dev_image_denoise, pixelcount * sizeof(glm::vec3));
  hipMemset(dev_image_denoise, 0, pixelcount * sizeof(glm::vec3));

  hipMalloc(&dev_image_denoise_tmp, pixelcount * sizeof(glm::vec3));
  hipMemset(dev_image_denoise_tmp, 0, pixelcount * sizeof(glm::vec3));

  hipMalloc(&dev_image_gBuffer, pixelcount * sizeof(glm::vec3));
  hipMemset(dev_image_gBuffer, 0, pixelcount * sizeof(glm::vec3));

  checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
  hipFree(dev_image);  // no-op if dev_image is null
  hipFree(dev_paths);
  hipFree(dev_geoms);
  hipFree(dev_materials);
  hipFree(dev_intersections);
  hipFree(dev_gBuffer);
  // TODO: clean up any extra device memory you created

  hipFree(dev_image_denoise);
  hipFree(dev_image_denoise_tmp);
  hipFree(dev_image_gBuffer);

  checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < cam.resolution.x && y < cam.resolution.y) {
    int index = x + (y * cam.resolution.x);
    PathSegment& segment = pathSegments[index];

    segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

    segment.ray.direction = glm::normalize(cam.view
      - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
      - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
    );

    segment.pixelIndex = index;
    segment.remainingBounces = traceDepth;
  }
}

__global__ void computeIntersections(
  int depth
  , int num_paths
  , PathSegment* pathSegments
  , Geom* geoms
  , int geoms_size
  , ShadeableIntersection* intersections
)
{
  int path_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (path_index < num_paths)
  {
    PathSegment pathSegment = pathSegments[path_index];

    float t;
    glm::vec3 intersect_point;
    glm::vec3 normal;
    float t_min = FLT_MAX;
    int hit_geom_index = -1;
    bool outside = true;

    glm::vec3 tmp_intersect;
    glm::vec3 tmp_normal;

    // naive parse through global geoms

    for (int i = 0; i < geoms_size; i++)
    {
      Geom& geom = geoms[i];

      if (geom.type == CUBE)
      {
        t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
      }
      else if (geom.type == SPHERE)
      {
        t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
      }

      // Compute the minimum t from the intersection tests to determine what
      // scene geometry object was hit first.
      if (t > 0.0f && t_min > t)
      {
        t_min = t;
        hit_geom_index = i;
        intersect_point = tmp_intersect;
        normal = tmp_normal;
      }
    }

    if (hit_geom_index == -1)
    {
      intersections[path_index].t = -1.0f;
    }
    else
    {
      //The ray hits something
      intersections[path_index].t = t_min;
      intersections[path_index].materialId = geoms[hit_geom_index].materialid;
      intersections[path_index].surfaceNormal = normal;
    }
  }
}

__global__ void shadeSimpleMaterials(
  int iter
  , int num_paths
  , ShadeableIntersection* shadeableIntersections
  , PathSegment* pathSegments
  , Material* materials
)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment segment = pathSegments[idx];
    if (segment.remainingBounces == 0) {
      return;
    }

    if (intersection.t > 0.0f) { // if the intersection exists...
      segment.remainingBounces--;
      // Set up the RNG
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, segment.remainingBounces);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        segment.color *= (materialColor * material.emittance);
        segment.remainingBounces = 0;
      }
      else {
        segment.color *= materialColor;
        glm::vec3 intersectPos = intersection.t * segment.ray.direction + segment.ray.origin;
        scatterRay(segment, intersectPos, intersection.surfaceNormal, material, rng);
      }
      // If there was no intersection, color the ray black.
      // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
      // used for opacity, in which case they can indicate "no opacity".
      // This can be useful for post-processing and image compositing.
    }
    else {
      segment.color = glm::vec3(0.0f);
      segment.remainingBounces = 0;
    }

    pathSegments[idx] = segment;
  }
}

__global__ void generateGBuffer(
  int num_paths,
  ShadeableIntersection* shadeableIntersections,
  PathSegment* pathSegments,
  GBufferPixel* gBuffer) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    gBuffer[idx].t = shadeableIntersections[idx].t;
    gBuffer[idx].normal = shadeableIntersections[idx].surfaceNormal;
    gBuffer[idx].pos = getPointOnRay(pathSegments[idx].ray, shadeableIntersections[idx].t);
  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < nPaths)
  {
    PathSegment iterationPath = iterationPaths[index];
    image[iterationPath.pixelIndex] += iterationPath.color;
  }
}

__global__ void denoiseATour(const Camera cam, const int stepWidth, const float c_phi, const float p_phi, const float n_phi,
  const glm::vec3* image, glm::vec3* imageDenoise, const GBufferPixel* gBuffer)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < cam.resolution.x && y < cam.resolution.y) {

    const float gaussian[9] = { 0.00390625, 0.015625, 0.0234375, 0.015625, 0.0625, 0.09375, 0.0234375, 0.09375, 0.140625 };

    glm::vec3 sum = glm::vec3(0.f);
    float cum_w = 0.f;

    int index = x + (y * cam.resolution.x);

    glm::vec3 cval = image[index];
    glm::vec3 pval = gBuffer[index].pos;
    glm::vec3 nval = gBuffer[index].normal;

    for (int j = -2; j <= 2; j++) {
      for (int i = -2; i <= 2; i++) {

        int uvX = min(max(int(x + i * stepWidth), 0), cam.resolution.x - 1);
        int uvY = min(max(int(y + j * stepWidth), 0), cam.resolution.y - 1);

        if (uvX < 0 || uvX >= cam.resolution.x) continue;
        if (uvY < 0 || uvY >= cam.resolution.y) continue;

        float kernelValue = gaussian[-abs(i) + 2 + (-abs(j) + 2) * 3];

        int itmp = uvX + cam.resolution.x * uvY;

        glm::vec3 ctmp = image[itmp];
        glm::vec3 t = cval - ctmp;
        float dist2 = glm::dot(t, t);
        float c_w = glm::min(std::exp(-(dist2) / (c_phi + EPSILON)), 1.f);

        t = nval - gBuffer[itmp].normal;
        dist2 = glm::max(glm::dot(t, t) / (stepWidth * stepWidth), 0.f);
        float n_w = glm::min(std::exp(-(dist2) / (n_phi + EPSILON)), 1.f);

        t = pval - gBuffer[itmp].pos;
        dist2 = glm::dot(t, t);
        float p_w = glm::min(std::exp(-(dist2) / (p_phi + EPSILON)), 1.f);

        float weight = c_w * p_w * n_w;
        sum += ctmp * weight * kernelValue;
        cum_w += weight * kernelValue;
      }
    }

    imageDenoise[index] = sum / cum_w;
  }
}

__global__ void diffuseImage(int nPaths, int iter, glm::vec3* image) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < nPaths)
  {
    glm::vec3 color = image[index];
    color.r /= iter;
    color.b /= iter;
    color.g /= iter;

    image[index] = color;
  }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter, bool isLast) {
  const int traceDepth = hst_scene->state.traceDepth;
  const Camera& cam = hst_scene->state.camera;
  const int pixelcount = cam.resolution.x * cam.resolution.y;

  // 2D block for generating ray from camera
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
    (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
    (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  // 1D block for path tracing
  const int blockSize1d = 128;

  ///////////////////////////////////////////////////////////////////////////

  // Pathtracing Recap:
  // * Initialize array of path rays (using rays that come out of the camera)
  //   * You can pass the Camera object to that kernel.
  //   * Each path ray must carry at minimum a (ray, color) pair,
  //   * where color starts as the multiplicative identity, white = (1, 1, 1).
  //   * This has already been done for you.
  // * NEW: For the first depth, generate geometry buffers (gbuffers)
  // * For each depth:
  //   * Compute an intersection in the scene for each path ray.
  //     A very naive version of this has been implemented for you, but feel
  //     free to add more primitives and/or a better algorithm.
  //     Currently, intersection distance is recorded as a parametric distance,
  //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
  //     * Color is attenuated (multiplied) by reflections off of any object
  //   * Stream compact away all of the terminated paths.
  //     You may use either your implementation or `thrust::remove_if` or its
  //     cousins.
  //     * Note that you can't really use a 2D kernel launch any more - switch
  //       to 1D.
  //   * Shade the rays that intersected something or didn't bottom out.
  //     That is, color the ray by performing a color computation according
  //     to the shader, then generate a new ray to continue the ray path.
  //     We recommend just updating the ray's PathSegment in place.
  //     Note that this step may come before or after stream compaction,
  //     since some shaders you write may also cause a path to terminate.
  // * Finally:
  //     * if not denoising, add this iteration's results to the image
  //     * TODO: if denoising, run kernels that take both the raw pathtraced result and the gbuffer, and put the result in the "pbo" from opengl

  generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
  checkCUDAError("generate camera ray");

  int depth = 0;
  PathSegment* dev_path_end = dev_paths + pixelcount;
  int num_paths = dev_path_end - dev_paths;

  // --- PathSegment Tracing Stage ---
  // Shoot ray into scene, bounce between objects, push shading chunks

  // Empty gbuffer
  hipMemset(dev_gBuffer, 0, pixelcount * sizeof(GBufferPixel));

  // clean shading chunks
  hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

  bool iterationComplete = false;
  while (!iterationComplete) {

    // tracing
    dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
    computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
      depth
      , num_paths
      , dev_paths
      , dev_geoms
      , hst_scene->geoms.size()
      , dev_intersections
      );
    checkCUDAError("trace one bounce");
    hipDeviceSynchronize();

    if (depth == 0) {
      generateGBuffer << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_intersections, dev_paths, dev_gBuffer);
    }

    depth++;

    shadeSimpleMaterials << <numblocksPathSegmentTracing, blockSize1d >> > (
      iter,
      num_paths,
      dev_intersections,
      dev_paths,
      dev_materials
      );
    iterationComplete = depth == traceDepth;
  }

  // Assemble this iteration and apply it to the image
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
  finalGather << <numBlocksPixels, blockSize1d >> > (num_paths, dev_image, dev_paths);

  ///////////////////////////////////////////////////////////////////////////

#if DENOISE

  if (isLast) {
    hipMemcpy(dev_image_denoise_tmp, dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

    diffuseImage << <numBlocksPixels, blockSize1d >> > (pixelcount, iter, dev_image_denoise_tmp);

    for (int stepWidth = 1; stepWidth * 4 <= ui_filterSize; stepWidth <<= 1) {
      denoiseATour << <blocksPerGrid2d, blockSize2d >> > (cam, stepWidth, ui_colorWeight, ui_positionWeight, ui_normalWeight,
        dev_image_denoise_tmp, dev_image_denoise, dev_gBuffer);

      std::swap(dev_image_denoise_tmp, dev_image_denoise);
    }
    std::swap(dev_image_denoise_tmp, dev_image_denoise);
  }
#endif

  // CHECKITOUT: use dev_image as reference if you want to implement saving denoised images.
  // Otherwise, screenshots are also acceptable.
  // Retrieve image from GPU
  hipMemcpy(hst_scene->state.image.data(), dev_image,
    pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

  checkCUDAError("pathtrace");
}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something that you can visualize for debugging.
void showGBuffer(uchar4* pbo, int iter, int type) {
  const Camera& cam = hst_scene->state.camera;
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
    (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
    (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
  gbufferToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, dev_image_gBuffer, cam.resolution, dev_gBuffer, type);

  hipMemcpy(hst_scene->state.image.data(), dev_image_gBuffer,
    cam.resolution.x * cam.resolution.y * sizeof(glm::vec3), hipMemcpyDeviceToHost);

  for (int x = 0; x < cam.resolution.x; x++) {
    for (int y = 0; y < cam.resolution.y; y++) {
      int index = x + (y * cam.resolution.x);
      hst_scene->state.image[index] *= iter;
    }
  }
}

void showImage(uchar4* pbo, int iter) {
  const Camera& cam = hst_scene->state.camera;
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
    (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
    (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  // Send results to OpenGL buffer for rendering
  sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);
}

void showImageDenoise(uchar4* pbo, int iter) {
  const Camera& cam = hst_scene->state.camera;
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
    (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
    (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  // Send results to OpenGL buffer for rendering
  sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, 1, dev_image_denoise);

  hipMemcpy(hst_scene->state.image.data(), dev_image_denoise,
    cam.resolution.x * cam.resolution.y * sizeof(glm::vec3), hipMemcpyDeviceToHost);

  for (int x = 0; x < cam.resolution.x; x++) {
    for (int y = 0; y < cam.resolution.y; y++) {
      int index = x + (y * cam.resolution.x);
      hst_scene->state.image[index] *= iter;
    }
  }
}