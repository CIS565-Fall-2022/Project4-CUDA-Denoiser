#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/partition.h>
#include <thrust/extrema.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#include <>

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}
	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

///////////////////////////////////////
__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void writeToImage(glm::ivec2 resolution,
	int iter, glm::vec3* image, PathSegment* paths
#if _ADAPTIVE_DEBUG_
	, PathSegment* s
#endif
	) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		PathSegment& path = paths[index];

		// contribution based on iter is inherently a bit scuffed
		glm::vec3 pix((image[index] * (float)(iter - 1) + path.color) / (float)iter);
#if _ADAPTIVE_DEBUG_
		// a inverse heatmap to exaggerate greys (not a lot of them though)
		//pix = glm::vec3((_MIN_SPP_ + 1) / (float)paths[index].spp);
		// linear ver. note that likelihood of termination plays a role in heat map.
		pix = glm::vec3((float)paths[index].spp / (float)s->spp);
		image[index] = pix;
#endif
#if _ADAPTIVE_SAMPLING_
		if (!path.skip) {
			image[index] = pix;
			if (path.terminate) {
				path.spp += 1;
				path.colorSum += path.color;
				path.magColorSumSq += glm::length2(path.color);
				if (_MIN_SPP_ < path.spp) {
					// mean and var per color
					float mean2 = glm::length2(path.colorSum / (float)path.spp);
					float variance = path.magColorSumSq / (float)path.spp - mean2;
					if (variance < _PIX_COV_TO_SKIP_) { // pixelIndex covariance or low  enough to assume OK
						path.skip = true; // skip on all future iterations
					}
				}
			}
		}
#else
		image[index] = pix;
#endif
	}
}

// generating kernel to create discrete wavelet transform
__device__ constexpr float generatingKern[5] = { 1.f / 16, 1.f / 4, 3.f / 8, 1.f / 4, 1.f / 16 };
__global__ void aTrousDenoise(glm::ivec2 resolution, glm::vec3* image, GBufferPixel *gBuffer,
	int stride, float colorWeight, float posnWeight, float normWeight) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		// A'Trous Denoising
		glm::vec3 sum(0.f);
		float totalWeight = 0.f;
#pragma unroll
		for (int i = -2; i <= 2; ++i) {
			for (int j = -2; j <= 2; ++j) {
				// Sample surrounding content; stride for empty btwn filter entries
				int sampleIndex = index + (i + resolution.x * j) * stride;
				if (0 <= sampleIndex && sampleIndex < resolution.x * resolution.y) {
					glm::vec3 diff = image[index] - image[sampleIndex];
					float dist = glm::dot(diff, diff);
					float cW = glm::min(glm::exp(-dist / colorWeight), 1.f);

					diff = gBuffer[index].norm - gBuffer[sampleIndex].norm;
					dist = glm::max(glm::dot(diff, diff) / (stride * stride), 0.f);
					float nW = glm::min(glm::exp(-dist / normWeight), 1.f);

					diff = gBuffer[index].posn - gBuffer[sampleIndex].posn;
					dist = glm::dot(diff, diff);
					float pW = glm::min(glm::exp(-dist / posnWeight), 1.f);

					float weight = cW * nW * pW *
						generatingKern[i + 2] * generatingKern[j + 2];
					sum += image[sampleIndex] * weight;
					totalWeight += weight;
				}
			}
		}
		image[index] = sum / totalWeight;
		// End A'Trous Denoising
	}
}


__global__ void bufToPBO(uchar4* pbo, glm::ivec2 resolution, glm::vec3* buffer) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		// Each thread writes one pixel location in the texture (textel)

		pbo[index].w = 0;
		pbo[index].x = glm::clamp((int)(buffer[index].x * 255.0), 0, 255);
		pbo[index].y = glm::clamp((int)(buffer[index].y * 255.0), 0, 255);
		pbo[index].z = glm::clamp((int)(buffer[index].z * 255.0), 0, 255);
	}
}

__global__ void gBufferFetchAttrib(glm::ivec2 resolution, GBufferPixel* gBuffer,
	int bit, glm::vec3* buffer) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = bit ?
			gBuffer[index].posn * 0.1f : //scale down posn, o/w wrapping is intense
			gBuffer[index].norm;
		pix = glm::abs(pix);

		buffer[index] = pix;
	}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static glm::vec3* dev_temp = NULL; // for misc. things like saving image
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static GBufferPixel* dev_gBuffer = NULL;
#if _CACHE_FIRST_BOUNCE_
static ShadeableIntersection* dev_first_isecs = NULL;
#endif

void InitDataContainer(GuiDataContainer* imGuiData) { guiData = imGuiData; }

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_temp, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

#if _CACHE_FIRST_BOUNCE_
	hipMalloc(&dev_first_isecs, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_first_isecs, 0, pixelcount * sizeof(ShadeableIntersection));
#endif
	hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_temp);
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	hipFree(dev_gBuffer);
#if _CACHE_FIRST_BOUNCE_
	hipFree(dev_first_isecs);
#endif
	checkCUDAError("pathtraceFree");
}

// From PBR Book, 13.6.2
// Concentrically maps pts from the [-1, 1) square to the disk centered at 0, 0.
__device__ glm::vec2 sampleDisk(thrust::default_random_engine &rng) {
	thrust::uniform_real_distribution<float> u01(-1, 1);
	glm::vec2 t(u01(rng), u01(rng));
	if (t.x == 0.f && t.y == 0.f) {
		return t; // glm::vec2(0.f);
	}
	float theta, r;
	if (std::abs(t.x) > std::abs(t.y)) {
		r = t.x;
		theta = PI / 4.f * (t.y / t.x);
	}
	else {
		r = t.y;
		theta = PI / 2.f - PI / 4.f * (t.x / t.y);
	}
	return r * glm::vec2(cos(theta), sin(theta));
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

#if _ADAPTIVE_SAMPLING_
		if (segment.skip) { // dont do anything for thisone
			segment.remainingBounces = 0;
			return;
		}
#endif
		segment.terminate = false;
		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);

		glm::vec2 jitter(0.f);
		// do not alias or motion blur if caching first bounce for obvious reasons
#if _CACHE_FIRST_BOUNCE_ 
#else
		if (cam.antialias) {
			const float stochasticWeight = 0.5f; // stddev
			thrust::normal_distribution<float> norm(0.f, stochasticWeight);
			jitter = glm::vec2(norm(rng), norm(rng));
		}

		thrust::uniform_real_distribution<float> u01(0, 1);
		segment.ray.time = u01(rng);
#endif
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + jitter.x)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f + jitter.y)
		);

		// Depth of field effects, from PBR book, 6.2.3
		if (cam.focalDist > 0 && cam.lensRad > 0) {
			// Set the origin ray to random place on lens (lens as disk).
			glm::vec2 lensRand = cam.lensRad * sampleDisk(rng);
			segment.ray.origin += glm::vec3(lensRand.x, lensRand.y, 0);

			glm::vec3 pFocus = 
				segment.ray.direction * cam.focalDist / std::abs(segment.ray.direction.z);
			segment.ray.direction = glm::normalize(pFocus - glm::vec3(lensRand.x, lensRand.y, 0));
		}

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
__global__ void computeIntersections(
	int depth,
	int num_paths,
	PathSegment* pathSegments,
	Geom* geoms,
	int geoms_size,
	ShadeableIntersection* intersections) {
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_index < num_paths) {
		PathSegment pathSegment = pathSegments[path_index];
#if _ADAPTIVE_SAMPLING_
		if (pathSegment.skip) { return; }
#endif
		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;
		bool tmp_outside;

		// naive parse through global geoms
		for (int i = 0; i < geoms_size; i++) {
			Geom& geom = geoms[i];
#if _CACHE_FIRST_BOUNCE_
			t = intersectionTest(geom.type, geom.transform, geom.inverseTransform, geom.invTranspose,
				pathSegment.ray, tmp_intersect, tmp_normal, tmp_outside);
#else
			// motion blur stuff
			if (geom.velocity != glm::vec3(0.f)) {
				glm::mat4 transform = dev_buildTransformationMatrix(
					geom.translation + pathSegment.ray.time * geom.velocity, geom.rotation, geom.scale);
				t = intersectionTest(
					geom.type, transform, glm::inverse(transform), glm::inverseTranspose(transform),
					pathSegment.ray, tmp_intersect, tmp_normal, tmp_outside);
			}
			else {
				t = intersectionTest(geom.type, geom.transform, geom.inverseTransform, geom.invTranspose,
					pathSegment.ray, tmp_intersect, tmp_normal, tmp_outside);
			}
#endif		
			// Compute minimum t from intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t) {
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
				outside = tmp_outside;
			}
		}

		if (hit_geom_index == -1) {
			intersections[path_index].t = -1.0f;
		}
		else {
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
			intersections[path_index].outside = outside;
		}
	}
}

// Shade path segments based on intersections, generate new rays w/ BSDF.
__global__ void shadeMaterial (
	int iter,
	int num_paths,
	ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
	Material* materials,
	int depth) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
#if _STREAM_COMPACTION_
#else // below skips if .skip = true (see generateRay)
	if (pathSegments[idx].remainingBounces == 0) { return; }
#endif
	if (idx < num_paths) {
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
			Material material = materials[intersection.materialId];
			if (material.emittance > 0.0f) { // material is bright; "light" the ray
				pathSegments[idx].terminate = true;
				pathSegments[idx].color *= (material.color * material.emittance);
				pathSegments[idx].remainingBounces = 0; // terminate if hit light
			} else {
				// BSDF evaluation: in-place assigns new direction + color for ray
				scatterRay(pathSegments[idx],
					getPointOnRay(pathSegments[idx].ray, intersection.t),
					intersection.surfaceNormal,
					intersection.outside,
					material,
					makeSeededRandomEngine(iter, idx, depth));
				if (--pathSegments[idx].remainingBounces == 0) {
					pathSegments[idx].color = BACKGROUND_COLOR;
				}
			}
		}
		else {
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
			pathSegments[idx].color = BACKGROUND_COLOR;
			pathSegments[idx].remainingBounces = 0; // terminate if hit nothing
		}
	}
}

__global__ void generateGBuffer(
	int num_paths,
	ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
	GBufferPixel* gBuffer) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths) {
		gBuffer[idx].norm = shadeableIntersections[idx].surfaceNormal;
		gBuffer[idx].posn = pathSegments[idx].ray.origin + 
			shadeableIntersections[idx].t * pathSegments[idx].ray.direction;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);
	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////
	// --- Start generate array of path rays (that come out of the camera)
	//   * Each path ray color starts as white = (1, 1, 1).
	generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");
	// --- End generate rays
	
	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks
	// Empty gbuffer
	hipMemset(dev_gBuffer, 0, pixelcount * sizeof(GBufferPixel));
	// clean shading chunks
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
	
	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;
	bool iterationComplete = false;
	while (!iterationComplete) {
		// --- start compute intersection ---
	//   * Compute an intersection in the scene for each path ray.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
#if _CACHE_FIRST_BOUNCE_
		if (iter > 1 && depth == 0) {
			hipMemcpy(dev_intersections,
				dev_first_isecs,
				pixelcount * sizeof(ShadeableIntersection),
				hipMemcpyDeviceToDevice);
		} else {
#endif
		computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
			depth,
			num_paths,
			dev_paths,
			dev_geoms,
			hst_scene->geoms.size(),
			dev_intersections);
		checkCUDAError("trace one bounce");
#if _CACHE_FIRST_BOUNCE_
		}
		if (iter == 1 && depth == 0) {
			hipMemcpy(dev_first_isecs,
				dev_intersections,
				pixelcount * sizeof(ShadeableIntersection),
				hipMemcpyDeviceToDevice);
		}
#endif
		hipDeviceSynchronize();
		if (depth == 0) {
			generateGBuffer<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_intersections, dev_paths, dev_gBuffer);
		}

		depth++;
		// --- end compute intersection --- 

#if _GROUP_RAYS_BY_MATERIAL_
		thrust::sort_by_key(thrust::device,
			dev_intersections, dev_intersections + num_paths,
			dev_paths,
			compare_intersection_mat());
#endif
		// can't use a 2D kernel launch any more - switch to 1D.
		// --- begin Shading Stage ---
		shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials,
			depth);
		// --- end shading stage
		
		// --- begin stream compaction for all of the terminated paths
		//	   determine if iterations should end (ie all paths done)
#if _STREAM_COMPACTION_
		dev_path_end = thrust::partition(thrust::device,
			dev_paths, dev_path_end,
			partition_terminated_paths()); // overloaded struct; lambdas need special compilation flag
		num_paths = dev_path_end - dev_paths;
		iterationComplete = (num_paths == 0);
#else
		iterationComplete = (depth == traceDepth);
#endif
		// --- end stream compaction

		if (guiData != NULL) {
			guiData->TracedDepth = depth;
		}
	}

#if _ADAPTIVE_DEBUG_
	PathSegment* s = thrust::max_element(thrust::device, dev_paths, dev_paths + pixelcount, compare_path_spp());
	writeToImage<<<blocksPerGrid2d, blockSize2d>>>(cam.resolution, iter, dev_image, dev_paths, s);
#else 
	writeToImage<<<blocksPerGrid2d, blockSize2d>>>(cam.resolution, iter, dev_image, dev_paths);
#endif
}

// bit = 0 for norm, 1 for posn. better as enum
void showGBuffer(uchar4* pbo, int bit, bool save) {
	const Camera& cam = hst_scene->state.camera;
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	gBufferFetchAttrib<<<blocksPerGrid2d, blockSize2d>>>
		(cam.resolution, dev_gBuffer, bit, dev_temp);

	if (!save) {
		bufToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, dev_temp);
	} else {
		bufToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, dev_temp);
		const int pixelcount = cam.resolution.x * cam.resolution.y;
		hipMemcpy(hst_scene->state.image.data(), dev_temp,
			pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
	}
}

// timing for analysis stuff
#if _TIME_ATROUS_DENOISER_
#include <chrono>
#endif

void showImage(uchar4* pbo, bool denoise, int filterSize, float colorWeight, float posWeight, float normWeight, bool save) {
	const Camera& cam = hst_scene->state.camera;
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// if denoising, use dev_temp for stuff
	if (denoise) {
		hipMemcpy(dev_temp, dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
		
#if _TIME_ATROUS_DENOISER_
		auto start = chrono::high_resolution_clock::now();
#endif
		for (int f = 0; f < filterSize; ++f) {
			aTrousDenoise<<<blocksPerGrid2d, blockSize2d>>>(cam.resolution, dev_temp, dev_gBuffer,
				1 << f, colorWeight, posWeight, normWeight);
		}
#if _TIME_ATROUS_DENOISER_
		auto stop = chrono::high_resolution_clock::now();
		auto duration = chrono::duration_cast<chrono::microseconds>(stop - start);
		cout << duration.count() << endl;
#endif
		bufToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, dev_temp);
		if (save) {
			hipMemcpy(hst_scene->state.image.data(), dev_temp, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
		}
	} else {
		bufToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, dev_image);
		if (save) {
			hipMemcpy(hst_scene->state.image.data(), dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
		}
	}
}