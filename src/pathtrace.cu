#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <chrono>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define SHOW_GBUFFER_NORMALS 0
#define SHOW_GBUFFER_POS 1

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__device__ uchar4 vec3ToColor(glm::vec3 v) {
  glm::vec3 col = glm::clamp(glm::abs(256.f * v), 0.f, 255.f);
  return make_uchar4(col.x, col.y, col.z, 0);
}

__global__ void gbufferToPBO(uchar4* pbo, glm::ivec2 resolution, GBufferPixel* gBuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);

#if SHOW_GBUFFER_NORMALS
        pbo[index] = vec3ToColor(gBuffer[index].normal);
#elif SHOW_GBUFFER_POS
        // scale down positions
        pbo[index] = vec3ToColor(gBuffer[index].position * 0.1f);
#endif
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static GBufferPixel* dev_gBuffer = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static glm::vec3* dev_image_denoised_in = NULL; // ping pong
static glm::vec3* dev_image_denoised_out = NULL;
static glm::ivec2 *dev_offset = NULL;
static float *dev_kernel = NULL;

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

    // TODO: initialize any extra device memeory you need
    hipMalloc(&dev_image_denoised_in, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image_denoised_in, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_image_denoised_out, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image_denoised_out, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_offset, 25 * sizeof(glm::ivec2));
    glm::ivec2 offset[25];
    for (int i = 0, int y = 0; y < 5; ++y) { // read array from left to right, top to bottom
      for (int x = 0; x < 5; ++x) {
        offset[i++] = glm::ivec2(x - 2, y - 2);
      }
    }
    hipMemcpy(dev_offset, offset, 25 * sizeof(glm::ivec2), hipMemcpyHostToDevice);

    hipMalloc(&dev_kernel, 25 * sizeof(float));
    float kernel[25] =
    { 1.f / 256, 1.f / 64, 3.f / 128, 1.f / 64, 1.f / 256,
      1.f / 64, 1.f / 16, 3.f / 32, 1.f / 16, 1.f / 64,
      3.f / 128, 3.f / 32, 9.f / 64, 3.f / 32, 3.f / 128,
      1.f / 64, 1.f / 16, 3.f / 32, 1.f / 16, 1.f / 64,
      1.f / 256, 1.f / 64, 3.f / 128, 1.f / 64, 1.f / 256 };
    hipMemcpy(dev_kernel, kernel, 25 * sizeof(float), hipMemcpyHostToDevice);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    hipFree(dev_gBuffer);
    // TODO: clean up any extra device memory you created
    hipFree(dev_image_denoised_in);
    hipFree(dev_image_denoised_out);
    hipFree(dev_offset);
    hipFree(dev_kernel);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void shadeSimpleMaterials (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment segment = pathSegments[idx];
    if (segment.remainingBounces == 0) {
      return;
    }

    if (intersection.t > 0.0f) { // if the intersection exists...
      segment.remainingBounces--;
      // Set up the RNG
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, segment.remainingBounces);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        segment.color *= (materialColor * material.emittance);
        segment.remainingBounces = 0;
      }
      else {
        segment.color *= materialColor;
        glm::vec3 intersectPos = intersection.t * segment.ray.direction + segment.ray.origin;
        scatterRay(segment, intersectPos, intersection.surfaceNormal, material, rng);
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      segment.color = glm::vec3(0.0f);
      segment.remainingBounces = 0;
    }

    pathSegments[idx] = segment;
  }
}

__global__ void generateGBuffer (
  int num_paths,
  ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
  GBufferPixel* gBuffer) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    auto& intersect = shadeableIntersections[idx];
    gBuffer[idx].normal = intersect.surfaceNormal;

    if (intersect.t < 0) {
      // Position doesn't matter too much since the colour is black anyway
      gBuffer[idx].position = glm::vec3(0);
    }
    else {
      auto& ray = pathSegments[idx].ray;
      gBuffer[idx].position = ray.origin + ray.direction * intersect.t;
    }
  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Pathtracing Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * NEW: For the first depth, generate geometry buffers (gbuffers)
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally:
    //     * if not denoising, add this iteration's results to the image
    //     * TODO: if denoising, run kernels that take both the raw pathtraced result and the gbuffer, and put the result in the "pbo" from opengl

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

  // Empty gbuffer
  hipMemset(dev_gBuffer, 0, pixelcount * sizeof(GBufferPixel));

	// clean shading chunks
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

  bool iterationComplete = false;
	while (!iterationComplete) {

	// tracing
	dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
	computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>> (
		depth
		, num_paths
		, dev_paths
		, dev_geoms
		, hst_scene->geoms.size()
		, dev_intersections
		);
	checkCUDAError("trace one bounce");
	hipDeviceSynchronize();

  if (depth == 0 && iter == 1) {
    auto start = std::chrono::system_clock::now();

    generateGBuffer << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_intersections, dev_paths, dev_gBuffer);

    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = end - start;
    std::cout << "G-buffer generation run-time (seconds): " << elapsed_seconds.count() << std::endl;
  }

	depth++;

  shadeSimpleMaterials<<<numblocksPathSegmentTracing, blockSize1d>>> (
    iter,
    num_paths,
    dev_intersections,
    dev_paths,
    dev_materials
  );
  iterationComplete = depth == traceDepth;
	}

  // Assemble this iteration and apply it to the image
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // CHECKITOUT: use dev_image as reference if you want to implement saving denoised images.
    // Otherwise, screenshots are also acceptable.
    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something that you can visualize for debugging.
void showGBuffer(uchar4* pbo) {
    const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
    gbufferToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, dev_gBuffer);
}

void showImage(uchar4* pbo, int iter) {
const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
}

__global__ void kernInitDenoiseBuffer(glm::vec3* image, glm::ivec2 resolution, float pathtraceIter, glm::vec3* image_denoised) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  if (!(x < resolution.x && y < resolution.y)) {
    return;
  }
  int index = x + (y * resolution.x);
  image_denoised[index] = image[index] / pathtraceIter;
}

__device__ float getWeight(glm::vec3 v1, glm::vec3 v2, float sigma) {
  glm::vec3 t = v1 - v2;
  float dist_squared = glm::max(glm::dot(t, t), 0.0f);
  return glm::min(exp(-dist_squared / (sigma * sigma)), 1.0f);
}

__global__ void kernDenoise(
  glm::ivec2 resolution,
  GBufferPixel *gBuffer, 
  int stepWidth,
  float *kernel,
  glm::ivec2 *offset,
  float colorWeight,
  float normalWeight,
  float positionWeight,
  glm::vec3 *image_denoised_in,
  glm::vec3 *image_denoised_out
) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x >= resolution.x || y >= resolution.y) {
    return;
  }

  int index = x + (y * resolution.x);

  auto &color = image_denoised_in[index];
  auto& position = gBuffer[index].position;
  auto& normal = gBuffer[index].normal;

  float cum_w = 0.0f;
  glm::vec3 sum(0.f);

  for (int i = 0; i < 25; ++i) {
    glm::ivec2 neighbourIdx = glm::ivec2(x, y) + offset[i] * stepWidth;

    if (neighbourIdx.x >= 0 && neighbourIdx.x < resolution.x
      && neighbourIdx.y >= 0 && neighbourIdx.y < resolution.y) {

      int n = neighbourIdx.x + (neighbourIdx.y * resolution.x);

      auto& neighbourColor = image_denoised_in[n];
      auto& neighbourPos = gBuffer[n].position;
      auto& neighbourNorm = gBuffer[n].normal;

      float c_w = getWeight(color, neighbourColor, colorWeight);
      float p_w = getWeight(position, neighbourPos, positionWeight);
      float n_w = getWeight(normal, neighbourNorm, normalWeight);

      float weight = c_w * n_w * p_w;
      sum += kernel[i] * weight * neighbourColor;
      cum_w += kernel[i] * weight;
    }
  }

  image_denoised_out[index] = sum / cum_w;
}

void denoiseAndWriteToPbo(
  uchar4* pbo,
  int pathtraceIter,
  int filterSize,
  float colorWeight,
  float normalWeight,
  float positionWeight,
  glm::ivec2 blockSize
) {
  const Camera& cam = hst_scene->state.camera;
  const dim3 blockSize2d(blockSize.x, blockSize.y);
  const dim3 blocksPerGrid2d(
    (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
    (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  kernInitDenoiseBuffer << <blocksPerGrid2d, blockSize2d >> > (dev_image, cam.resolution, pathtraceIter, dev_image_denoised_in);

  // filter size is size of window on the last iteration
  int numDenoiseIters = glm::log2(filterSize / 5);
  int stepWidth = 1;

  for (int i = 0; i < numDenoiseIters; ++i) {
    kernDenoise << <blocksPerGrid2d, blockSize2d >> > (
      cam.resolution,
      dev_gBuffer,
      stepWidth,
      dev_kernel,
      dev_offset,
      colorWeight,
      normalWeight,
      positionWeight,
      dev_image_denoised_in,
      dev_image_denoised_out);

    // filter doubles every iter
    stepWidth = stepWidth << 2;
    // At each pass we set sigma rt = 2^{-i} * sigma_rt
    // allowing for smaller illumination variations to be smoothed
    colorWeight = colorWeight / stepWidth;

    std::swap(dev_image_denoised_in, dev_image_denoised_out); // most updated version is _in now
  }
  sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, 1, dev_image_denoised_in);

  hipMemcpy(hst_scene->state.image.data(), dev_image_denoised_in,
    cam.resolution.x * cam.resolution.y * sizeof(glm::vec3), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
}
