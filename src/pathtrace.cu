#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__global__ void gbufferToPBO(uchar4* pbo, glm::ivec2 resolution, GBufferPixel* gBuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        float timeToIntersect = gBuffer[index].t * 256.0;
        glm::vec3 normal = gBuffer[index].normal;
        glm::ivec3 normalColor;
        normalColor.x = glm::clamp((int)abs((normal.x * 255.0)), 0, 255);
        normalColor.y = glm::clamp((int)abs((normal.y * 255.0)), 0, 255);
        normalColor.z = glm::clamp((int)abs((normal.z * 255.0)), 0, 255);
        pbo[index].w = 0;
        //pbo[index].x = timeToIntersect;
        //pbo[index].y = timeToIntersect;
        //pbo[index].z = timeToIntersect;
        float x = gBuffer[index].normal.x;
        float y = gBuffer[index].normal.y;
        float z = gBuffer[index].normal.z;
        pbo[index].x = normalColor.x;
        pbo[index].y = normalColor.y;
        pbo[index].z = normalColor.z;
        //glm::vec3 position = gBuffer[index].position;
        //glm::ivec3 positionColor;
        //positionColor.x = glm::clamp((int)abs((position.x * 32.0)), 0, 255);
        //positionColor.y = glm::clamp((int)abs((position.y * 32.0)), 0, 255);
        //positionColor.z = glm::clamp((int)abs((position.z * 32.0)), 0, 255);
        //pbo[index].x = positionColor.x;
        //pbo[index].y = positionColor.y;
        //pbo[index].z = positionColor.z;

    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static GBufferPixel* dev_gBuffer = NULL;
static float host_kernel[25] = { 1 / 256 ,  1 / 64 , 3 / 128 , 1 / 64 , 1 / 256,
    1 / 64 , 1 / 16 , 3 / 32 , 1 / 16 , 1 / 64,
    3 / 128 , 3 / 32 , 9 / 64 , 3 / 32 , 3 / 128,
    1 / 64 , 1 / 16 , 3 / 32 , 1 / 16 , 1 / 64,
    1 / 256 , 1 / 64 , 3 / 128 , 1 / 64 , 1 / 256 };
static float* dev_kernel = NULL;
static glm::vec3* dev_image_denoise = NULL;
static glm::vec3* dev_image_denoise_ping_pong = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

    // TODO: initialize any extra device memeory you need
    hipMalloc(&dev_kernel, 25 * sizeof(float));
    hipMemcpy(dev_kernel, host_kernel, 25 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&dev_image_denoise, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image_denoise, 0, pixelcount * sizeof(glm::vec3));
    hipMalloc(&dev_image_denoise_ping_pong, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image_denoise_ping_pong, 0, pixelcount * sizeof(glm::vec3));
    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    hipFree(dev_gBuffer);
    hipFree(dev_kernel);
    hipFree(dev_image_denoise);
    hipFree(dev_image_denoise_ping_pong);
    // TODO: clean up any extra device memory you created

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void shadeSimpleMaterials (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment segment = pathSegments[idx];
    if (segment.remainingBounces == 0) {
      return;
    }

    if (intersection.t > 0.0f) { // if the intersection exists...
      segment.remainingBounces--;
      // Set up the RNG
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, segment.remainingBounces);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        segment.color *= (materialColor * material.emittance);
        segment.remainingBounces = 0;
      }
      else {
        segment.color *= materialColor;
        glm::vec3 intersectPos = intersection.t * segment.ray.direction + segment.ray.origin;
        scatterRay(segment, intersectPos, intersection.surfaceNormal, material, rng);
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      segment.color = glm::vec3(0.0f);
      segment.remainingBounces = 0;
    }

    pathSegments[idx] = segment;
  }
}

__global__ void generateGBuffer (
  int num_paths,
  ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
  GBufferPixel* gBuffer) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    
    gBuffer[idx].t = shadeableIntersections[idx].t;
    if (gBuffer[idx].t != -1.0f)
    {
        gBuffer[idx].normal = shadeableIntersections[idx].surfaceNormal;
        gBuffer[idx].position = getPointOnRay(pathSegments[idx].ray, shadeableIntersections[idx].t);
    }
    else
    {
        gBuffer[idx].normal = glm::vec3(0.0f);
        gBuffer[idx].position = glm::vec3(0.0f);
    }

  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}
__global__ void denoise(GBufferPixel* gBuffer, Camera cam, glm::vec3* image, float* dumb_kernel, glm::vec3* image_denoise, int level, float colorWeight, float normalWeight, float positionWeight)
{
    //TODO: Test fastest way
    float kernel_25[25] = { 1.0f / 256.0f ,  1.0f / 64.0f , 3.0f / 128.0f , 1.0f / 64.0f , 1.0f / 256.0f,
    1.0f / 64.0f , 1.0f / 16.0f , 3.0f / 32.0f , 1.0f / 16.0f , 1.0f / 64.0f,
    3.0f / 128.0f , 3.0f / 32.0f , 9.0f / 64.0f , 3.0f / 32.0f , 3.0f / 128.0f,
    1.0f / 64.0f , 1.0f / 16.0f , 3.0f / 32.0f , 1.0f / 16.0f , 1.0f / 64.0f,
    1.0f / 256.0f , 1.0f / 64.0f , 3.0f / 128.0f , 1 / 64.0f , 1.0f / 256.0f };
    float kernel[5] = { 1.f / 16.f, 1.f / 4.f, 3.f / 8.f, 1.f / 4.f , 1.f / 16.f };
    int x_o = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y_o = (blockIdx.y * blockDim.y) + threadIdx.y;
    int origin_index = x_o + (y_o * cam.resolution.x);
    float cum_weight = 0;
    glm::vec3 sum;
    if (x_o < cam.resolution.x && y_o < cam.resolution.y)
    {
#pragma unroll
        for (int j = 0; j < 5; j++)
        {
#pragma unroll
            for (int i = 0; i < 5; i++)
            {
                int x = x_o + (i - 2) * (1 << level);
                int y = y_o + (j - 2) * (1 << level);
                // x = glm::clamp(x, 0, cam.resolution.x - 1);
                // y = glm::clamp(y, 0, cam.resolution.y - 1);
                if (x < cam.resolution.x && y < cam.resolution.y && (x >= 0) && (y) >= 0)
                {
                    //Cap weights at 1.0 per paper
                    int index = (x + (y * cam.resolution.x));
                    float kernel_weight = kernel[i] * kernel[j];
                    glm::vec3 t = image[origin_index] - image_denoise[index];
                    float c_w = glm::min(glm::exp(-glm::dot(t, t) / (colorWeight)),1.0f);
                    t = gBuffer[origin_index].normal - gBuffer[index].normal;
                    //Need to update based on step size?
                    float n_w = glm::min(glm::exp(-glm::dot(t, t) / (normalWeight )), 1.0f);
                    t = gBuffer[origin_index].position - gBuffer[index].position;
                    float p_w = glm::min(glm::exp(-glm::dot(t, t) / (positionWeight)), 1.0f);
                    float weight = c_w * n_w * p_w;
                    sum += (kernel_weight * image_denoise[index] * weight);
                    cum_weight += kernel_weight * weight;
                }
            }

        }

        image[origin_index] = sum/cum_weight;
    }
}
/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Pathtracing Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * NEW: For the first depth, generate geometry buffers (gbuffers)
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally:
    //     * if not denoising, add this iteration's results to the image
    //     * TODO: if denoising, run kernels that take both the raw pathtraced result and the gbuffer, and put the result in the "pbo" from opengl

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

  // Empty gbuffer
  hipMemset(dev_gBuffer, 0, pixelcount * sizeof(GBufferPixel));

	// clean shading chunks
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

  bool iterationComplete = false;
	while (!iterationComplete) {

	// tracing
	dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
	computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>> (
		depth
		, num_paths
		, dev_paths
		, dev_geoms
		, hst_scene->geoms.size()
		, dev_intersections
		);
	checkCUDAError("trace one bounce");
	hipDeviceSynchronize();

  if (depth == 0) {
    generateGBuffer<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_intersections, dev_paths, dev_gBuffer);
  }

	depth++;

  shadeSimpleMaterials<<<numblocksPathSegmentTracing, blockSize1d>>> (
    iter,
    num_paths,
    dev_intersections,
    dev_paths,
    dev_materials
  );
  iterationComplete = depth == traceDepth;
	}

  // Assemble this iteration and apply it to the image
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////
    //if (iter == 10)
    //{
    //    hipMemcpy(dev_image_denoise, dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
    //    for (int sweep = 0; sweep < 4; sweep++)
    //    {
    //        std::swap(dev_image, dev_image_denoise);
    //        //hipMemcpy(dev_image_denoise, dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
    //        denoise << < blocksPerGrid2d, blockSize2d >> > (cam, num_paths, dev_image, dev_paths, dev_kernel, dev_image_denoise, sweep);
    //        hipDeviceSynchronize();    
    //    }
    //    
    //}
    // CHECKITOUT: use dev_image as reference if you want to implement saving denoised images.
    // Otherwise, screenshots are also acceptable.
    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
void runDenoiser(int filterSize, float colorWeight, float normalWeight, float positionWeight)
{
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);
    //Avoid negative values
    int denoise_interations = filterSize < 5 ? 0 : log2((filterSize / 5.0f));
    std::cout << denoise_interations << std::endl;;
    hipMemcpy(dev_image_denoise, dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
    hipMemcpy(dev_image_denoise_ping_pong, dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
    int num_paths = 0;
    for (int sweep = 0; sweep < denoise_interations; sweep++)
    {
        std::swap(dev_image_denoise, dev_image_denoise_ping_pong);
        denoise << < blocksPerGrid2d, blockSize2d >> > (dev_gBuffer, cam, dev_image_denoise, dev_kernel, dev_image_denoise_ping_pong, sweep, colorWeight * colorWeight, normalWeight * normalWeight, positionWeight * positionWeight);
        hipDeviceSynchronize();
    }
    checkCUDAError("denoise");

}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something that you can visualize for debugging.
void showGBuffer(uchar4* pbo) {
    const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
    gbufferToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, dev_gBuffer);
}



void showImage(uchar4* pbo, int iter) {
const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
}

void showDenoisedImage(uchar4* pbo, int iter) {
    const Camera& cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image_denoise);
}
