#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1
#define DENOISE_ITERATIONS 5

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

__global__ void denoiseBasicBlur(glm::vec3* denoisedImage, glm::ivec2 resolution,
    int iter, int stepsize, glm::vec3* image, GBufferPixel* dev_gBuffer,
    float* dev_kernel, glm::ivec2* dev_offset) {
   int x = (blockIdx.x * blockDim.x) + threadIdx.x;
   int y = (blockIdx.y * blockDim.y) + threadIdx.y;

   if (x < resolution.x && y < resolution.y) {
       int index = x + (y * resolution.x); 

       glm::vec3 sum = glm::vec3(0.f, 0.f, 0.f);
       for (int i = 0; i < 25; i++) {
           glm::ivec2 offset = dev_offset[i] * stepsize;
           glm::ivec2 uv = glm::ivec2(x, y) + offset;

           // Clamp indices to image width and height
           uv = glm::clamp(uv, glm::ivec2(0, 0), glm::ivec2(resolution.x - 1, resolution.y - 1));

           // Apply kernel
           glm::vec3 col = image[uv.x + resolution.x * uv.y];
           sum += col * dev_kernel[i];
       }
       
       // Write color to OpenGL PBO
       denoisedImage[index] = sum;
   }
}

__global__ void denoiseWeighted(glm::vec3* denoisedImage, glm::ivec2 resolution,
    int iter, int stepsize, float sigma_col, float sigma_norm, float sigma_pos,
    glm::vec3* image, GBufferPixel* dev_gBuffer,
    float* dev_kernel, glm::ivec2* dev_offset) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);

        // Read G-Buffer values for current pixel
        glm::vec3 col = image[index];
        glm::vec3 norm = dev_gBuffer[index].normal;
        glm::vec3 pos = dev_gBuffer[index].position;

        // Sum to accumulate color
        glm::vec3 sum = glm::vec3(0.f, 0.f, 0.f);
        float sum_weights = 0.f;
        
        glm::vec3 test_col;

        for (int i = 0; i < 25; i++) {
            // Find pixel in image at desired offset (clamp to image boundaries)
            glm::ivec2 offset = dev_offset[i] * stepsize;
            glm::ivec2 uv = glm::ivec2(x, y) + offset;
            uv = clamp(uv, glm::ivec2(0, 0), glm::ivec2(resolution.x - 1, resolution.y - 1));

            // Find difference in color between current and neighboring pixel
            glm::vec3 col_n = image[uv.x + resolution.x * uv.y];
            glm::vec3 col_diff = col - col_n;
            float dist2 = glm::dot(col_diff, col_diff);
            float col_w = glm::min(glm::exp(-(dist2) / (sigma_col * sigma_col)), 1.f);

            // Normal
            glm::vec3 norm_n = dev_gBuffer[uv.x + resolution.x * uv.y].normal;
            glm::vec3 norm_diff = norm - norm_n;
            dist2 = glm::max(glm::dot(norm_diff, norm_diff) / (float)(stepsize * stepsize), 0.f);
            float norm_w = glm::min(glm::exp(-(dist2) / (sigma_norm * sigma_norm)), 1.f);

            // Position
            glm::vec3 pos_n = dev_gBuffer[uv.x + resolution.x * uv.y].position;
            glm::vec3 pos_diff = pos - pos_n;
            dist2 = glm::dot(pos_diff, pos_diff);
            float pos_w = glm::min(glm::exp(-(dist2) / (sigma_pos * sigma_pos)), 1.f);

            // Calculate weighting
            float weight = col_w * norm_w * pos_w;
            sum += col_n * weight * dev_kernel[i];
            sum_weights += weight * dev_kernel[i];
        }

        // Write color to OpenGL PBO
        denoisedImage[index] = sum / sum_weights;
    }

}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__global__ void gbufferToPBO(uchar4* pbo, int ui_currentBuffer, glm::ivec2 resolution, GBufferPixel* gBuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        float timeToIntersect = gBuffer[index].t * 256.0;
        glm::vec3 position = 0.1f * abs(gBuffer[index].position);
        glm::vec3 normal = 0.5f * (gBuffer[index].normal + glm::vec3(1.f, 1.f, 1.f));

        if (ui_currentBuffer == 0) {
            pbo[index].w = 0;
            pbo[index].x = timeToIntersect;
            pbo[index].y = timeToIntersect;
            pbo[index].z = timeToIntersect;
        }
        else if (ui_currentBuffer == 1) {
            pbo[index].w = 0;
            pbo[index].x = normal.x * 255.0;
            pbo[index].y = normal.y * 255.0;
            pbo[index].z = normal.z * 255.0;
        }
        else {
            pbo[index].w = 0;
            pbo[index].x = position.x * 256.0;
            pbo[index].y = position.y * 256.0;
            pbo[index].z = position.z * 256.0;
        }
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static glm::vec3* dev_denoised_image_in = NULL;
static glm::vec3* dev_denoised_image_out = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static GBufferPixel* dev_gBuffer = NULL;
static float* dev_kernel = NULL;
static glm::ivec2* dev_offsets = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

glm::ivec2 offsets[25];

float kernel[25] = { 1.f / 256.f, 1.f / 64.f, 3.f / 128.f, 1.f / 64.f, 1.f / 256.f,
                     1.f / 64.f, 1.f / 16.f, 3.f / 32.f, 1.f / 16.f, 1 / 64.f,
                     3.f / 128.f, 3.f / 32.f, 9.f / 64.f, 3.f / 32.f, 3.f / 128.f,
                     1.f / 64.f, 1.f / 16.f, 3.f / 32.f, 1.f / 16.f, 1 / 64.f,
                     1.f / 256.f, 1.f / 64.f, 3.f / 128.f, 1.f / 64.f, 1.f / 256.f };

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // Fill offset array
    int count = 0;
    for (int j = -2; j <= 2; ++j) {
        for (int i = -2; i <= 2; ++i) {
            offsets[count] = glm::ivec2(i, j);
            //std::cout << "(" << count << "): " << "(" << offsets[count].x << ", " << offsets[count].y << ")" << std::endl;
            ++count;
        }
    }

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    // To store intermediate results after denoising
    hipMalloc(&dev_denoised_image_in, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_denoised_image_in, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_denoised_image_out, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_denoised_image_out, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

    // TODO: initialize any extra device memeory you need
    hipMalloc(&dev_kernel, 25 * sizeof(float));
    hipMemcpy(dev_kernel, kernel, 25 * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&dev_offsets, 25 * sizeof(glm::ivec2));
    hipMemcpy(dev_offsets, offsets, 25 * sizeof(glm::ivec2), hipMemcpyHostToDevice);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    hipFree(dev_gBuffer);
    // TODO: clean up any extra device memory you created
    hipFree(dev_denoised_image_in);
    hipFree(dev_denoised_image_out);
    hipFree(dev_kernel);
    hipFree(dev_offsets);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void shadeSimpleMaterials (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment segment = pathSegments[idx];
    if (segment.remainingBounces == 0) {
      return;
    }

    if (intersection.t > 0.0f) { // if the intersection exists...
      segment.remainingBounces--;
      // Set up the RNG
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, segment.remainingBounces);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        segment.color *= (materialColor * material.emittance);
        segment.remainingBounces = 0;
      }
      else {
        segment.color *= materialColor;
        glm::vec3 intersectPos = intersection.t * segment.ray.direction + segment.ray.origin;
        scatterRay(segment, intersectPos, intersection.surfaceNormal, material, rng);
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      segment.color = glm::vec3(0.0f);
      segment.remainingBounces = 0;
    }

    pathSegments[idx] = segment;
  }
}

__global__ void generateGBuffer (
  int num_paths,
  ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
  GBufferPixel* gBuffer) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    gBuffer[idx].t = shadeableIntersections[idx].t * 0.05f;
    gBuffer[idx].position = getPointOnRay(pathSegments[idx].ray, shadeableIntersections[idx].t);
    gBuffer[idx].normal = shadeableIntersections[idx].surfaceNormal;
  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Pathtracing Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * NEW: For the first depth, generate geometry buffers (gbuffers)
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally:
    //     * if not denoising, add this iteration's results to the image
    //     * TODO: if denoising, run kernels that take both the raw pathtraced result and the gbuffer, and put the result in the "pbo" from opengl

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

    // Empty gbuffer
    hipMemset(dev_gBuffer, 0, pixelcount * sizeof(GBufferPixel));

	// clean shading chunks
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    bool iterationComplete = false;
	while (!iterationComplete) {

	    // tracing
	    dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
	    computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>> (
		    depth
		    , num_paths
		    , dev_paths
		    , dev_geoms
		    , hst_scene->geoms.size()
		    , dev_intersections
		    );
	    checkCUDAError("trace one bounce");
	    hipDeviceSynchronize();

        // For first depth, generate gBuffer
        if (depth == 0) {
            generateGBuffer<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_intersections, dev_paths, dev_gBuffer);
        }

	    depth++;

        shadeSimpleMaterials<<<numblocksPathSegmentTracing, blockSize1d>>> (
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials
        );
        iterationComplete = depth == traceDepth;
	}

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // CHECKITOUT: use dev_image as reference if you want to implement saving denoised images.
    // Otherwise, screenshots are also acceptable.
    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something that you can visualize for debugging.
void showGBuffer(uchar4* pbo, int ui_currentBuffer) {
    const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
    gbufferToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, ui_currentBuffer, cam.resolution, dev_gBuffer);
}

void denoise(uchar4* pbo, int iter, float sigma_col, float sigma_norm, float sigma_pos) {
    const Camera& cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Copy initial image input data
    int pixelcount = cam.resolution.x * cam.resolution.y;
    hipMemcpy(dev_denoised_image_in, dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

    //std::cout << "in denoise" << std::endl;

    // Denoise image
    int stepsize = 1;
    float sigma_col_div = sigma_col;
    for (int i = 0; i < DENOISE_ITERATIONS; ++i) {
        denoiseWeighted << <blocksPerGrid2d, blockSize2d >> > (dev_denoised_image_out, cam.resolution, iter, stepsize,
            sigma_col_div, sigma_norm, sigma_pos,
            dev_denoised_image_in, dev_gBuffer, dev_kernel, dev_offsets);
        if (i != DENOISE_ITERATIONS - 1) {
            std::swap(dev_denoised_image_in, dev_denoised_image_out);
        }
        stepsize *= 2;
        sigma_col_div /= (float)stepsize;
    }

    // Send results to OpenGL buffer for rendering
    sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_denoised_image_out);
}

void showImage(uchar4* pbo, int iter) {
const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
}
