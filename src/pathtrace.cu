#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// TODO Modify this so that we can viz different parts of the gbuffer
__global__ void gbufferToPBO(uchar4* pbo, glm::ivec2 resolution, GBufferPixel* gBuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        //float timeToIntersect = gBuffer[index].t * 256.0;
        //glm::vec3 viz = (gBuffer[index].normal + glm::vec3(1.0)) / glm::vec3(2.0) * glm::vec3(255.0);
        if (gBuffer[index].t > 0) {
            float position_range = 25.f;
            glm::vec3 viz = (glm::clamp(gBuffer[index].position, glm::vec3(-position_range), glm::vec3(position_range)) + position_range) / (position_range * 2.f) * 255.f;
            pbo[index].w = 0;
            pbo[index].x = viz.r;
            pbo[index].y = viz.g;
            pbo[index].z = viz.b;
        }
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static GBufferPixel* dev_gBuffer = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

static float* dev_filter = NULL;
static glm::vec2* dev_offsets = NULL;
// Kernel/Filter from https://www.eso.org/sci/software/esomidas/doc/user/18NOV/volb/node317.html
const float filter[25] = { 1.0 / 256.0, 1.0 / 64.0,  3.0 / 128.0, 1.0 / 64.0, 1.0 / 256.0,
                           1.0 / 64.0,  1.0 / 16.0,  3.0 / 32.0,  1.0 / 16.0, 1.0 / 64.0,
                           3.0 / 128.0, 3.0 / 32.0,  9.0 / 64.0,  3.0 / 32.0, 3.0 / 128.0,
                           1.0 / 64.0,  1.0 / 16.0,  3.0 / 32.0,  1.0 / 16.0, 1.0 / 64.0,
                           1.0 / 256.0, 1.0 / 64.0,  3.0 / 128.0, 1.0 / 64.0, 1.0 / 256.0,
};
// Offsets (x, y)
const glm::vec2 offsets[25] = { glm::vec2(-2, -2), glm::vec2(-1, -2), glm::vec2(0, -2), glm::vec2(1, -2), glm::vec2(2, -2),
                                glm::vec2(-2, -1), glm::vec2(-1, -1), glm::vec2(0, -1), glm::vec2(1, -1), glm::vec2(2, -1),
                                glm::vec2(-2, 0),  glm::vec2(-1, 0),  glm::vec2(0, 0),  glm::vec2(1, 0),  glm::vec2(2, 0),
                                glm::vec2(-2, 1),  glm::vec2(-1, 1),  glm::vec2(0, 1),  glm::vec2(1, 1),  glm::vec2(2, 1),
                                glm::vec2(-2, 2),  glm::vec2(-1, 2),  glm::vec2(0, 2),  glm::vec2(1, 2),  glm::vec2(2, 2),
};
// Temp denoise output buffer for ping ponging
static glm::vec3* dev_denoise_in = NULL;
static glm::vec3* dev_denoise_out = NULL;
// Stuff for timing
static hipEvent_t startTime = NULL;
static hipEvent_t endTime = NULL; 

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

    // TODO: initialize any extra device memeory you need
    hipMalloc(&dev_filter, 25 * sizeof(float));
    hipMemcpy(dev_filter, &filter, 25 * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&dev_offsets, 25 * sizeof(glm::vec2));
    hipMemcpy(dev_offsets, &offsets, 25 * sizeof(glm::vec2), hipMemcpyHostToDevice);

    hipMalloc(&dev_denoise_in, pixelcount * sizeof(glm::vec3));
    hipMalloc(&dev_denoise_out, pixelcount * sizeof(glm::vec3));

    hipEventCreate(&startTime);
    hipEventCreate(&endTime);
    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    hipFree(dev_gBuffer);
    // TODO: clean up any extra device memory you created
    hipFree(dev_filter);
    hipFree(dev_offsets);
    hipFree(dev_denoise_in);
    hipFree(dev_denoise_out);

    if (startTime != NULL) {
        hipEventDestroy(startTime);
    }
    if (endTime != NULL) {
        hipEventDestroy(endTime);
    }
    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void shadeSimpleMaterials (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment segment = pathSegments[idx];
    if (segment.remainingBounces == 0) {
      return;
    }

    if (intersection.t > 0.0f) { // if the intersection exists...
      segment.remainingBounces--;
      // Set up the RNG
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, segment.remainingBounces);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        segment.color *= (materialColor * material.emittance);
        segment.remainingBounces = 0;
      }
      else {
        segment.color *= materialColor;
        glm::vec3 intersectPos = intersection.t * segment.ray.direction + segment.ray.origin;
        scatterRay(segment, intersectPos, intersection.surfaceNormal, material, rng);
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      segment.color = glm::vec3(0.0f);
      segment.remainingBounces = 0;
    }

    pathSegments[idx] = segment;
  }
}


// TODO ADD NORMALS, XYZ to this
__global__ void generateGBuffer (
  int num_paths,
  ShadeableIntersection* shadeableIntersections,
  PathSegment* pathSegments,
  GBufferPixel* gBuffer) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    gBuffer[idx].t = shadeableIntersections[idx].t;
    gBuffer[idx].normal = shadeableIntersections[idx].surfaceNormal;
    gBuffer[idx].position = shadeableIntersections[idx].t * pathSegments[idx].ray.direction + pathSegments[idx].ray.origin;
  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Pathtracing Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * NEW: For the first depth, generate geometry buffers (gbuffers)
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally:
    //     * if not denoising, add this iteration's results to the image
    //     * TODO: if denoising, run kernels that take both the raw pathtraced result and the gbuffer, and put the result in the "pbo" from opengl

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

    // Empty gbuffer
    hipMemset(dev_gBuffer, 0, pixelcount * sizeof(GBufferPixel));

	// clean shading chunks
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    bool iterationComplete = false;
	while (!iterationComplete) {

	    // tracing
	    dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
	    computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>> (
		    depth
		    , num_paths
		    , dev_paths
		    , dev_geoms
		    , hst_scene->geoms.size()
		    , dev_intersections
		    );
	    checkCUDAError("trace one bounce");
	    hipDeviceSynchronize();

        if (depth == 0) {
            generateGBuffer<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_intersections, dev_paths, dev_gBuffer);
        }

	    depth++;

        shadeSimpleMaterials<<<numblocksPathSegmentTracing, blockSize1d>>> (
        iter,
        num_paths,
        dev_intersections,
        dev_paths,
        dev_materials
        );
        iterationComplete = depth == traceDepth;
	}

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // CHECKITOUT: use dev_image as reference if you want to implement saving denoised images.
    // Otherwise, screenshots are also acceptable.
    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something that you can visualize for debugging.
void showGBuffer(uchar4* pbo) {
    const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
    gbufferToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, dev_gBuffer);
}

void showImage(uchar4* pbo, int iter) {
    const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
}

__global__ void denoise(glm::vec3* dev_imageIn, glm::vec3* dev_imageOut, const int stepWidth, const glm::vec2 resolution,
                        const glm::vec2* dev_offsets, const float* dev_filter, const float colorSigma, const float normalSigma, 
                        const float positionSigma, const GBufferPixel* dev_gBuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int originalIndex = x + (y * resolution.x);
        
        //Center point values (current pixel)
        glm::vec3 originalColor = dev_imageIn[originalIndex];
        glm::vec3 originalNorm = dev_gBuffer[originalIndex].normal;
        glm::vec3 originalPos = dev_gBuffer[originalIndex].position;

        glm::vec3 sum = glm::vec3(0.0);
        float cumW = 0.0;

        for (int i = 0; i < 25; ++i) { // Get neighbors
            glm::vec2 neighbor_offset = dev_offsets[i] * glm::vec2(stepWidth);
            int neighborX = x + neighbor_offset.x;
            int neighborY = y + neighbor_offset.y;
            if (neighborX >= 0 && neighborX < resolution.x && neighborY >= 0 && neighborY < resolution.y) { // check bounds of image
                int neighborIndex = neighborX + (neighborY * resolution.x);

                glm::vec3 color = dev_imageIn[neighborIndex];
                float colorWeight = min(exp(-(glm::length2(originalColor - color)) / colorSigma), 1.f);
                
                glm::vec3 norm = dev_gBuffer[neighborIndex].normal;
                float normWeight = min(exp(-(max(glm::length2(originalNorm - norm) / (stepWidth * stepWidth), 0.f) / normalSigma)), 1.f);

                glm::vec3 pos = dev_gBuffer[neighborIndex].position;
                float posWeight = min(exp(-(glm::length2(originalPos - norm) / positionSigma)), 1.f);

                float weight = colorWeight * normWeight * posWeight;
                sum += color * weight * dev_filter[i];
                cumW += weight * dev_filter[i];
                //blurred_pix += dev_filter[i] * dev_imageIn[neighbor_index];
            }
        }
        dev_imageOut[originalIndex] = sum / cumW;
    }
}

void showDenoise(uchar4* pbo, int iter, const int filterSize, const float colorSigma, const float normalSigma, const float positionSigma) {
    const Camera& cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // Copy image to denoise buffer so it doesnt affect orignial image
    hipMemcpy(dev_denoise_in, dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
    const float squaredColor = pow(colorSigma, 2);
    const float squaredNormal = pow(normalSigma * .1, 2);
    const float squaredPos = pow(positionSigma, 2);
    int i = 0;
    float time;
    hipEventRecord(startTime);
    while (4 * (1 << i) + 1 < filterSize)  { // Multiple iterations of denoising
        int stepWidth = 1 << i;
        denoise << <blocksPerGrid2d, blockSize2d >> > (dev_denoise_in, dev_denoise_out, stepWidth, 
                                                        cam.resolution, dev_offsets, dev_filter,
                                                        squaredColor, squaredNormal, squaredPos, dev_gBuffer);
        hipDeviceSynchronize();
        //Ping pong buffers
        glm::vec3* temp = dev_denoise_in;
        dev_denoise_in = dev_denoise_out;
        dev_denoise_out = temp;
        ++i;
    }
    hipEventRecord(endTime);
    hipEventSynchronize(endTime);
    hipEventElapsedTime(&time, startTime, endTime);
    std::cout << "Time denoise: " << time << std::endl;
    sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_denoise_in);
}