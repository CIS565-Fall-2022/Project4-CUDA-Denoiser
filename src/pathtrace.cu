#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include <cmath>

#include <>

#define ERRORCHECK 1
#define SORT_MATERIAL 1
#define STREAM_COMPACTION 1
#define CACHE_INTERSECTION 1
#define ANTI_ALIASING 1
#define DOF 0
#define MOTION_BLUR_OBJECT 0
#define MOTION_BLUR_CAMERA 0
#define BOUNDING_BOX 1

#define GBUFFER_NORMAL 1
#define TIMER 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}
__host__ __device__ glm::vec2 squareToDiskUniform(thrust::default_random_engine& rng) {
	float M_PI = 3.1415926;

	thrust::uniform_real_distribution<float> u01(0, 1);

	glm::vec2 u = glm::vec2(u01(rng), u01(rng));
	glm::vec2 uOffset = 2.f * u - glm::vec2(1.f, 1.f);

	if (uOffset.x == 0 && uOffset.y == 0) {
		return glm::vec2(0.f, 0.f);
	}
	float theta, r;
	if (std::abs(uOffset.x) > std::abs(uOffset.y)) {
		r = uOffset.x;
		theta = M_PI / 4.f * (uOffset.y / uOffset.x);
	}
	else {
		r = uOffset.y;
		theta = M_PI / 2.f - M_PI / 4.f * (uOffset.x / uOffset.y);
	}
	return r * glm::vec2(std::cos(theta), std::sin(theta));
}
//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

__global__ void gbufferToPBO(uchar4* pbo, glm::ivec2 resolution, GBufferPixel* gBuffer) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		//float timeToIntersect = gBuffer[index].t * 256.0;

		//pbo[index].w = 0;
		//pbo[index].x = timeToIntersect;
		//pbo[index].y = timeToIntersect;
		//pbo[index].z = timeToIntersect;

		// normal
		//pbo[index].x = gBuffer[index].norm[0] * 256.0;
		//pbo[index].y = gBuffer[index].norm[1] * 256.0;
		//pbo[index].z = gBuffer[index].norm[2] * 256.0;
			
		glm::vec3 outputPosition = glm::normalize(gBuffer[index].posn);
		//glm::vec3 outputPosition = gBuffer[index].posn;

#ifdef GBUFFER_NORMAL
		pbo[index].x = glm::clamp((int)(abs(gBuffer[index].norm.x * 255.0)), 0, 255);
		pbo[index].y = glm::clamp((int)(abs(gBuffer[index].norm.y * 255.0)), 0, 255);
		pbo[index].z = glm::clamp((int)(abs(gBuffer[index].norm.z * 255.0)), 0, 255);
#else
		outputPosition *= 255.0;
		pbo[index].w = 0;
		pbo[index].x = abs(outputPosition.x);
		pbo[index].y = abs(outputPosition.y);
		pbo[index].z = abs(outputPosition.z);
	
#endif // GBUFFER_NORMAL

}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static ShadeableIntersection* dev_cache_intersections = NULL;
static TriangleGeom* dev_triangles = NULL;
static GBufferPixel* dev_gBuffer = NULL;
static glm::vec3* dev_denoise_image = NULL;
static glm::vec3* dev_denoise_temp = NULL;


#if TIMER
hipEvent_t start, stop;
float totaltime = 0.f;
#endif


void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
#if TIMER
	hipEventCreate(&start);
	hipEventCreate(&stop);
#endif

	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// TODO: initialize any extra device memeory you need
	hipMalloc(&dev_cache_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_cache_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_triangles, scene->meshGeoms.triangleGeoms.size() * sizeof(TriangleGeom));
	hipMemcpy(dev_triangles, scene->meshGeoms.triangleGeoms.data(), scene->meshGeoms.triangleGeoms.size() * sizeof(TriangleGeom), hipMemcpyHostToDevice);

	hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

	// denoise buffer
	hipMalloc(&dev_denoise_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_denoise_image, 0, pixelcount * sizeof(glm::vec3));
	hipMalloc(&dev_denoise_temp, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_denoise_temp, 0, pixelcount * sizeof(glm::vec3));
	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
#if TIMER
	if (start != NULL)
		hipEventDestroy(start);
	if (stop != NULL)
		hipEventDestroy(stop);
#endif

	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	// TODO: clean up any extra device memory you created
	hipFree(dev_cache_intersections);
	hipFree(dev_triangles);
	hipFree(dev_gBuffer);

	hipFree(dev_denoise_image);
	hipFree(dev_denoise_temp);

	checkCUDAError("pathtraceFree");
}

__global__ void kernBlur(Geom* geoms, int geoms_size, int num_paths, glm::vec3 moveDir, int iter) {
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_index < num_paths) {
		float dt = iter * 0.00001f;
		if (iter < 200 || iter > 800) {
			return;
		}
		for (int i = 0; i < geoms_size; ++i) {
			Geom& geom = geoms[i];
			if (geom.type == SPHERE) {
				geom.translation -= glm::clamp(moveDir * dt, glm::vec3(0.0f), moveDir);
				geom.transform[3] = glm::vec4(geom.translation, geom.transform[3].w);
				geom.inverseTransform = glm::inverse(geom.transform);
				geom.invTranspose = glm::transpose(geom.inverseTransform);
			}
		}
	}
}
/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		// TODO: implement antialiasing by jittering the ray
		//segment.ray.direction = glm::normalize(cam.view
		//	- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
		//	- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		//);

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);



#if ANTI_ALIASING
		thrust::uniform_real_distribution<float> u01(0, 1);
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + u01(rng))
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f +u01(rng))
		);
#else
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);
#endif

#if MOTION_BLUR_CAMERA
		float dt = iter * 0.1f;
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + 2.0f * dt* u01(rng))
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f + u01(rng))
		);
#else
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);
#endif
#if DOF
		// code from CIS 561
		cam.focalDistance = 10.f;
		cam.radius = 0.9f;

		glm::vec2 randomSample{ 0 };
		float ft = glm::abs((cam.focalDistance) / segment.ray.direction.z);

		// point of focus using original ray direction
		glm::vec3 pFocus = ft * segment.ray.direction;

		// sample a point on the lens
		glm::vec2 pLens = cam.radius * squareToDiskUniform(rng);


		segment.ray.origin += glm::vec3(pLens.x, pLens.y, 0.f);
		segment.ray.direction = glm::normalize(pFocus - glm::vec3(pLens.x, pLens.y, 0.f));

#endif

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}
__host__ __device__ bool isInBoundingBox(Geom& object,
	glm::vec3 min, glm::vec3 max, Ray& r) {
	Ray q;
	q.origin = multiplyMV(object.inverseTransform, glm::vec4(r.origin, 1.0f));
	q.direction = glm::normalize(multiplyMV(object.inverseTransform, glm::vec4(r.direction, 0.0f)));

	float tmin = FLT_MIN;
	float tmax = FLT_MAX;
	glm::vec3 tmin_n;
	glm::vec3 tmax_n;
	for (int xyz = 0; xyz < 3; ++xyz) {
		float qdxyz = q.direction[xyz];
		if (glm::abs(qdxyz) <= 0.00001f) {
			break;
		}
		float t1 = (min[xyz] - q.origin[xyz]) / qdxyz;
		float t2 = (max[xyz] - q.origin[xyz]) / qdxyz;
		float ta = glm::min(t1, t2);
		float tb = glm::max(t1, t2);
		glm::vec3 n;
		n[xyz] = t2 < t1 ? +1 : -1;
		if (ta > 0 && ta > tmin) {
			tmin = ta;
			tmin_n = n;
		}
		if (tb < tmax) {
			tmax = tb;
			tmax_n = n;
		}
		
	}

	if (tmax >= tmin && tmax > 0) {
		return true;
	}
	return false;
}
// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, Geom* geoms
	, int geoms_size
	, TriangleGeom* triangles
	, int triangles_size
	, glm::vec3 min
	, glm::vec3 max
	, ShadeableIntersection* intersections
	
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == MESH)
			{
				float z = FLT_MAX;
#if BOUNDING_BOX
				if (!isInBoundingBox(geom, min, max, pathSegment.ray)) {
					break;
				}
#endif
				for (int j = 0; j < triangles_size; j++) {
					TriangleGeom& tri = triangles[j];

					float triangle_inter = triangleInteractionTest(geom, pathSegment.ray, tmp_intersect,
						tri.v1, tri.v2, tri.v3, tri.n1, tri.n2, tri.n3, tmp_normal, outside);
					if (triangle_inter != -1) {
						z = glm::min(z, triangle_inter);
					}
				}
				t = z;

			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;

		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(1.0f, 1.0f, 0.0f));
				pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				//pathSegments[idx].color *= materialColor;

				pathSegments[idx].color *= u01(rng); // apply some noise because why not
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
		}
	}
}

__global__ void shadeBSDFMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;


	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (pathSegments[idx].remainingBounces == 0) { return; }

		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].remainingBounces = 0;

				pathSegments[idx].color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {

				glm::vec3 intersectPoint = getPointOnRay(pathSegments[idx].ray, intersection.t);

				scatterRay(pathSegments[idx], intersectPoint, intersection.surfaceNormal, material, rng);
				--pathSegments[idx].remainingBounces;
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			pathSegments[idx].remainingBounces = 0;

			pathSegments[idx].color = glm::vec3(0.0f);
		}
	}
}


__global__ void generateGBuffer(
	int num_paths,
	ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
	GBufferPixel* gBuffer) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		gBuffer[idx].norm = shadeableIntersections[idx].surfaceNormal;
		/*gBuffer[idx].posn = pathSegments[idx].ray.origin +
			shadeableIntersections[idx].t * pathSegments[idx].ray.direction;*/
		gBuffer[idx].posn = getPointOnRay(pathSegments[idx].ray, shadeableIntersections[idx].t);
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}
struct isPathCompleted 
{
	__host__ __device__
		bool operator()(const PathSegment& pathSegment) {
		return pathSegment.remainingBounces > 0;
	}
};

struct sortMaterialsID
{
	__host__ __device__ 
		bool operator()(ShadeableIntersection& intersect1, const ShadeableIntersection& intersect2)
	{
		return intersect1.materialId < intersect2.materialId;
	}
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	//while (!iterationComplete && depth < traceDepth) {
	hipMemset(dev_gBuffer, 0, pixelcount * sizeof(GBufferPixel));

	// clean shading chunks
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	//bool iterationComplete = false;
	while (!iterationComplete) {

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		std::cout << "depth: " << depth << "\n";
#if MOTION_BLUR_OBJECT
		glm::vec3 velocity = glm::vec3(-1e-5f, -1e-6f, 1e-2f);
		kernBlur << <numblocksPathSegmentTracing, blockSize1d >> > (dev_geoms, hst_scene->geoms.size(), num_paths, velocity, iter);
#endif //BLURGEOM

#if CACHE_INTERSECTION && !defined(ANTI_ALIASING)
		if (depth == 0 && iter == 1) {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_triangles
				, hst_scene->meshGeoms.triangleGeoms.size()
				, hst_scene->meshGeoms.min
				, hst_scene->meshGeoms.max
				, dev_intersections
				);
			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();
			hipMemcpy(dev_cache_intersections, dev_intersections,
				pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);

		}
		else if (depth == 0) {
			hipMemcpy(dev_intersections, dev_cache_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);

		}
		else {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_triangles
				, hst_scene->meshGeoms.triangleGeoms.size()
				, hst_scene->meshGeoms.min
				, hst_scene->meshGeoms.max
				, dev_intersections
				);
			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();
		}
#else
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, num_paths
			, dev_paths
			, dev_geoms
			, hst_scene->geoms.size()
			, dev_triangles
			, hst_scene->meshGeoms.triangleGeoms.size()
			, hst_scene->meshGeoms.min
			, hst_scene->meshGeoms.max
			, dev_intersections
			);
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		
#endif // CACHE_INTERSECTION

		if (depth == 0) {
			generateGBuffer << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_intersections, dev_paths, dev_gBuffer);
		}

		depth++;
		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.


#ifdef SORT_MATERIAL

			// sort by material type,  thrust::stable_sort_by_key(thrust::host, keys, keys + N, values, predicate);
			thrust::stable_sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, sortMaterialsID());

#endif // SORT_MATERIAL

		shadeBSDFMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials
			);
#ifdef STREAM_COMPACTION

		// stream compaction
		dev_path_end = thrust::stable_partition(thrust::device, dev_paths, dev_path_end, isPathCompleted());
		num_paths = dev_path_end - dev_paths;

		//std::cout << num_paths << "\n";
		iterationComplete = (num_paths == 0);
#else
		iterationComplete = (++depth >= traceDepth);
#endif // SORT_MATERIAL

		//std::cout << "num_paths: " << num_paths << std::endl;

		//iterationComplete = (++depth >= tracekDepth);

		if (guiData != NULL)
		{
			guiData->TracedDepth = depth;
		}
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_image, dev_paths);

	///////////////////////////////////////////////////////////////////////////

	// CHECKITOUT: use dev_image as reference if you want to implement saving denoised images.
	// Otherwise, screenshots are also acceptable.
	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something that you can visualize for debugging.
void showGBuffer(uchar4* pbo) {
	const Camera& cam = hst_scene->state.camera;
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
	gbufferToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, dev_gBuffer);
}

void showImage(uchar4* pbo, int iter) {
	const Camera& cam = hst_scene->state.camera;
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);
}

// adapted from slides code
__global__ void ATrousFilter(glm::ivec2 resolution, glm::vec3* inColorBuffer, glm::vec3* outFragBuffer,
	GBufferPixel* gbuffer, float c_phi, float n_phi, float p_phi, float stepwidth) {
	int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
	int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx_x >= resolution.x || idx_y >= resolution.y) {
		return;
	}

	glm::vec3 sum = glm::vec3(0.0f);
	glm::vec2 step = glm::vec2(1 / resolution.x, 1 / resolution.y);
	glm::vec3 kernel = glm::vec3(0.375f, 0.25f, 0.0625f);

	glm::vec3 cval = inColorBuffer[idx_y * resolution.x + idx_x];
	glm::vec3 nval = gbuffer[idx_y * resolution.x + idx_x].norm;
	glm::vec3 pval = gbuffer[idx_y * resolution.x + idx_x].posn;
	
	float cum_w = 0.f;

	for (int i = -2; i <= 2; i++) {
		for (int j = -2; j <= 2; j++) {
			glm::ivec2 uv = glm::clamp(glm::ivec2(idx_x + j * stepwidth, idx_y + i * stepwidth), glm::ivec2(0, 0), resolution);

			glm::vec3 ctmp = inColorBuffer[uv.x + uv.y * resolution.x];
			glm::vec3 t = cval - ctmp;
			float dist2 = glm::dot(t, t);
			float c_w = min(exp(-dist2 / (c_phi + 0.0001f)), 1.f);

			glm::vec3 ntmp = gbuffer[uv.x + uv.y * resolution.x].norm;
			t = nval - ntmp;
			dist2 = max(glm::dot(t, t) / (stepwidth * stepwidth), 0.f);
			float n_w = min(exp(-dist2 / (n_phi + 0.0001f)), 1.f);

			glm::vec3 ptmp = gbuffer[uv.x + uv.y * resolution.x].posn;
			t = pval - ptmp;
			dist2 = glm::dot(t, t);
			float p_w = min(exp(-dist2 / (p_phi + 0.0001f)), 1.f);

			float weight = c_w * n_w * p_w;

			float kernel2D = kernel[abs(i)] * kernel[abs(j)];

			sum += ctmp * weight * kernel2D;
			cum_w += weight * kernel2D;
		}
	}
	if (cum_w < 0.0001f) { return; }
	outFragBuffer[idx_y * resolution.x + idx_x] = sum / cum_w;
}
__global__ void denoiseInit(int iteration, glm::ivec2 resolution, glm::vec3* denoise1, glm::vec3* image)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.y && y < resolution.y) {
		int idx = x + resolution.x * y;
		glm::vec3 pixel = image[idx];

		denoise1[idx].x = pixel.x / iteration;
		denoise1[idx].y = pixel.y / iteration;
		denoise1[idx].z = pixel.z / iteration;
	}
}
void applyDenoise(float c_phi, float n_phi, float p_phi, float filtersize, int iter) {
	const Camera& cam = hst_scene->state.camera;
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

#if TIMER
	hipEventRecord(start);
#endif

	int atrou_iter = glm::floor(log2((filtersize - 5) / 4.f)) + 1;
	int pixelcount = cam.resolution.x * cam.resolution.y;
	denoiseInit << <blocksPerGrid2d, blockSize2d >> > (iter, cam.resolution, dev_denoise_image, dev_image);

	for (int i = 0; i < atrou_iter; i++) {
		float stepwidth = 1 << i;
		ATrousFilter << < blocksPerGrid2d, blockSize2d >> > (cam.resolution,
			dev_denoise_image, dev_denoise_temp, dev_gBuffer, c_phi, n_phi, p_phi, stepwidth);
		//Ping Pong
		glm::vec3* temp = dev_denoise_temp;
		dev_denoise_temp = dev_denoise_image;
		dev_denoise_image = temp;
	}

#if TIMER
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	totaltime += time;
	
	std::cout << "DENOISE TIME: " << totaltime << std::endl;
	
#endif

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_denoise_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
}

void showDenoiseBuffer(uchar4* pbo) {
	const Camera& cam = hst_scene->state.camera;
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, 1, dev_denoise_image);
}
void copyDevImage()
{
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;
	hipMemcpy(hst_scene->state.image.data(), dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
}